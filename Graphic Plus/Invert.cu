#include "hip/hip_runtime.h"  


static unsigned char *data = NULL;
static int N;

__global__ void Invert(unsigned char* img, bool invertTransparentPixel)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (invertTransparentPixel || img[4 * i + 3])
	{
		img[4 * i] = 255 - img[4 * i];
		img[4 * i + 1] = 255 - img[4 * i + 1];
		img[4 * i + 2] = 255 - img[4 * i + 2];
	}
}

extern "C" void addInvertKernel(unsigned *imageData, int *args)
{
	hipSetDevice(0);
	N = args[0] * args[1];
	if (data == NULL)
		hipMalloc((void**)&data, N * sizeof(unsigned));
	hipMemcpy(data, imageData, N * sizeof(unsigned), hipMemcpyHostToDevice);
	dim3 blockSize(args[0]);
	dim3 gridSize(args[1]);
	Invert <<<gridSize, blockSize >>> (data, bool(args[2]));
	hipDeviceSynchronize();
	hipMemcpy(imageData, data, N * sizeof(unsigned), hipMemcpyDeviceToHost);
}

extern "C" void freeInvertKernel()
{
	if (data != NULL)
		hipFree(data);
}

extern "C" void reMallocInvert(int width, int height)
{
	freeInvertKernel();

	hipMalloc((void**)&data, width * height * sizeof(unsigned));
}