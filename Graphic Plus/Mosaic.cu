#include "hip/hip_runtime.h"  

#include <cmath>

using std::ceilf;

static unsigned char *oriImg = NULL, *blurImg = NULL;

__global__ void MosaicSquare(
	unsigned char *const output,
	const unsigned char *const input,
	int rows,
	int cols,
	int size
	)
{
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	int pos = (r * cols + c) * 4;

	if ((r >= rows) || (c >= cols))
	{
		return;
	}

	int left = (c / size) * size;
	int right = min((c / size + 1) * size, cols);
	int top = (r / size) * size;
	int bottom = min((r / size + 1) * size, rows);
	int cpos = ((top + bottom) * cols / 2 + (left + right) / 2) * 4;

	output[pos] = input[cpos];
	output[pos + 1] = input[cpos + 1];
	output[pos + 2] = input[cpos + 2];
	output[pos + 3] = 255;
};

__global__ void MosaicCircle(
	unsigned char *const output,
	const unsigned char *const input,
	int rows,
	int cols,
	int size
	)
{
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	int pos = (r * cols + c) * 4;

	if ((r >= rows) || (c >= cols))
	{
		return;
	}

	int left = (c / size) * size;
	int right = min((c / size + 1) * size, cols);
	int top = (r / size) * size;
	int bottom = min((r / size + 1) * size, rows);
	int centc = (left + right) / 2;
	int centr = (top + bottom) / 2;
	int cpos = (centr * cols + centc) * 4;

	if ((r - centr) * (r - centr) + (c - centc) * (c - centc) <= (size / 2) * (size / 2))
	{
		output[pos] = input[cpos];
		output[pos + 1] = input[cpos + 1];
		output[pos + 2] = input[cpos + 2];
		output[pos + 3] = 255;
	}
	else
	{
		int num = 0, cr = 0, cc = 0;
		int red = 0, green = 0, blue = 0;
		for (int i = -8; i <= 8; i++)
		{
			for (int j = -8; j <= 8; j++)
			{
				cr = r + i;
				cc = c + j;
				if (cr < 0 || cr >= rows || cc < 0 || cc >= cols)
					continue;
				cpos = (cr * cols + cc) * 4;
				red += input[cpos];
				green += input[cpos + 1];
				blue += input[cpos + 2];
				num++;
			}
		}

		output[pos] = red / num;
		output[pos + 1] = green / num;
		output[pos + 2] = blue / num;
		output[pos + 3] = 255;
	}
}

extern "C" void freeMosaicKernel()
{
	if (oriImg != NULL)
		hipFree(oriImg);
	if (blurImg != NULL)
		hipFree(blurImg);
}

extern "C" void reMallocMosaic(int width, int height)
{
	if (oriImg != NULL)
		hipFree(oriImg);
	if (blurImg != NULL)
		hipFree(blurImg);

	hipMalloc((void**)&oriImg, width * height * sizeof(unsigned));
	hipMalloc((void**)&blurImg, width * height * sizeof(unsigned));
}

extern "C" void addMosaicKernel(unsigned *imageData, int *args)
{
	hipSetDevice(0);
	if (oriImg == NULL)
		reMallocMosaic(args[0], args[1]);

	hipMemcpy(oriImg, imageData, args[0] * args[1] * sizeof(unsigned), hipMemcpyHostToDevice);
	static const int BLOCK_WIDTH = 32;

	int x = static_cast<int>(ceilf(static_cast<float>(args[0]) / BLOCK_WIDTH));
	int y = static_cast<int>(ceilf(static_cast<float>(args[1]) / BLOCK_WIDTH));

	const dim3 grid(x, y, 1);
	const dim3 block(BLOCK_WIDTH, BLOCK_WIDTH, 1);

	switch (args[2])
	{
		case 0:
			MosaicSquare <<< grid, block >>> (blurImg, oriImg, args[1], args[0], args[3]);
			break;

		case 1:
			MosaicCircle <<< grid, block >>> (blurImg, oriImg, args[1], args[0], args[3]);
			break;
	}
	hipDeviceSynchronize();

	hipMemcpy(imageData, blurImg, args[0] * args[1] * sizeof(unsigned), hipMemcpyDeviceToHost);
}