#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"  
#include ""

unsigned *data = NULL;
int N;

__global__ void grayScale(unsigned *img, int N)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned gray = ((img[i] & 0xFFU) + ((img[i] & 0xFF00U) >> 8) + ((img[i] & 0xFF0000U) >> 16)) / 3;
	img[i] = 0xFF000000U | gray << 16 | gray << 8 | gray;
}

extern "C" void addKernel(unsigned *imageData, int width, int height)
{
	hipSetDevice(0);
	N = width * height;
	if(data == NULL)
		hipMalloc((void**)&data, N * sizeof(unsigned));
	hipMemcpy(data, imageData, N * sizeof(unsigned), hipMemcpyHostToDevice);
	dim3 blockSize(width);
    dim3 gridSize(height);
	grayScale <<<gridSize, blockSize>>> (data, N);
	hipDeviceSynchronize();
	hipMemcpy(imageData, data, N * sizeof(unsigned), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
}

extern "C" void freeKernel()
{
	if(data != NULL)
		hipFree(data);
}

extern "C" void reMalloc(int width, int height)
{
	if(data != NULL)
		hipFree(data);

	hipMalloc((void**)&data, width * height * sizeof(unsigned));
}