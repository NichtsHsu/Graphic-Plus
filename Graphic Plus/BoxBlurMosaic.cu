#include "hip/hip_runtime.h"  

#include <cmath>

using std::ceilf;

static unsigned char *oriImg = NULL, *blurImg = NULL;

__global__ void BoxBlurMosaic(
	unsigned char *const output,
	const unsigned char *const input,
	int rows,
	int cols,
	int blockWidth,
	int blockHieght
	)
{
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	int pos = (r * cols + c) * 4;

	if ((r >= rows) || (c >= cols))
	{
		return;
	}

	int left = (c / blockWidth) * blockWidth;
	int right = min((c / blockWidth + 1) * blockWidth, cols);
	int top = (r / blockHieght) * blockHieght;
	int bottom = min((r / blockHieght + 1) * blockHieght, rows);
	int num = 0, cr = 0, cc = 0, cpos;
	int red = 0, green = 0, blue = 0;
	for (int i = -2; i <= 2; i++)
	{
		for (int j = -2; j <= 2; j++)
		{
			cr = r + i;
			cc = c + j;
			if (cr < top || cr >= bottom || cc < left || cc >= right)
				continue;
			cpos = (cr * cols + cc) * 4;
			red += input[cpos];
			green += input[cpos + 1];
			blue += input[cpos + 2];
			num++;
		}
	}

	output[pos] = red / num;
	output[pos + 1] = green / num;
	output[pos + 2] = blue / num;
	output[pos + 3] = 255;
}

extern "C" void freeBoxBlurMosaicKernel()
{
	if (oriImg != NULL)
		hipFree(oriImg);
	if (blurImg != NULL)
		hipFree(blurImg);
}

extern "C" void reMallocBoxBlurMosaic(int width, int height)
{
	if (oriImg != NULL)
		hipFree(oriImg);
	if (blurImg != NULL)
		hipFree(blurImg);

	hipMalloc((void**)&oriImg, width * height * sizeof(unsigned));
	hipMalloc((void**)&blurImg, width * height * sizeof(unsigned));
}

extern "C" void addBoxBlurMosaicKernel(unsigned *imageData, int *args)
{
	hipSetDevice(0);
	if (oriImg == NULL)
		reMallocBoxBlurMosaic(args[0], args[1]);

	hipMemcpy(oriImg, imageData, args[0] * args[1] * sizeof(unsigned), hipMemcpyHostToDevice);
	static const int BLOCK_WIDTH = 32;

	int x = static_cast<int>(ceilf(static_cast<float>(args[0]) / BLOCK_WIDTH));
	int y = static_cast<int>(ceilf(static_cast<float>(args[1]) / BLOCK_WIDTH));

	const dim3 grid(x, y, 1);
	const dim3 block(BLOCK_WIDTH, BLOCK_WIDTH, 1);

	for (int i = 0; i < args[2]; i++)
	{
		BoxBlurMosaic <<< grid, block >>>(blurImg, oriImg, args[1], args[0], args[3], args[4]);
		hipMemcpy(oriImg, blurImg, args[0] * args[1] * sizeof(unsigned), hipMemcpyDeviceToDevice);
		hipDeviceSynchronize();
	}

	hipMemcpy(imageData, blurImg, args[0] * args[1] * sizeof(unsigned), hipMemcpyDeviceToHost);
}