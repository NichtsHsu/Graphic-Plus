#include "hip/hip_runtime.h"  


static unsigned *data = NULL;
static int N;

__global__ void grayScale(unsigned *img)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned gray = ((img[i] & 0xFFU) + ((img[i] & 0xFF00U) >> 8) + ((img[i] & 0xFF0000U) >> 16)) / 3;
	img[i] = 0xFF000000U | gray << 16 | gray << 8 | gray;
}

extern "C" void addGrayScaleKernel(unsigned *imageData, int *args)
{
	hipSetDevice(0);
	N = args[0] * args[1];
	if(data == NULL)
		hipMalloc((void**)&data, N * sizeof(unsigned));
	hipMemcpy(data, imageData, N * sizeof(unsigned), hipMemcpyHostToDevice);
	dim3 blockSize(args[0]);
    dim3 gridSize(args[1]);
	grayScale<<<gridSize, blockSize>>> (data);
	hipDeviceSynchronize();
	hipMemcpy(imageData, data, N * sizeof(unsigned), hipMemcpyDeviceToHost);
}

extern "C" void freeGrayScaleKernel()
{
	if(data != NULL)
		hipFree(data);
}

extern "C" void reMallocGrayScale(int width, int height)
{
	freeGrayScaleKernel();

	hipMalloc((void**)&data, width * height * sizeof(unsigned));
}